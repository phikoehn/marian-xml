#include "hip/hip_runtime.h"
#include "language_model.h"


LanguageModel::LanguageModel(const LM& lm)
: SourceIndependentScorer(), lm_(lm)
{}

void LanguageModel::Score(const State& in,
				   Prob& prob,
				   State& out) {

  const LMState& lmIn = in.get<LMState>();
  LMState& lmOut = out.get<LMState>();

  size_t rows = prob.dim(0);
  size_t cols = prob.Cols();

  std::vector<float> costs(rows * cols);
  const std::vector<KenlmState>& inStates = lmIn.GetStates();
  std::vector<KenlmState>& outStates = lmOut.GetStates();
  outStates.resize(rows * cols);

  for(size_t i = 0; i < inStates.size(); i++) {
	  KenlmState stateUnk;
	  float costUnk = lm_.Score(inStates[i], 0, stateUnk);
	  std::fill(costs.begin() + i * cols, costs.begin() + i * cols + cols, costUnk);
	  std::fill(outStates.begin() + i * cols, outStates.begin() + i * cols + cols, stateUnk);
  }

  {
	ThreadPool pool(God::Get<size_t>("kenlm-batch-threads"));
	size_t batchSize = God::Get<size_t>("kenlm-batch-size");
	for(size_t batchStart = 0; batchStart < lm_.size(); batchStart += batchSize) {
	  auto call = [batchStart, batchSize, cols, this, &costs, &inStates, &outStates] {
		size_t batchEnd = min(batchStart + batchSize, lm_.size());
		for(auto it = lm_.begin() + batchStart; it != lm_.begin() + batchEnd; ++it)
		  if(it->second < cols)
			for(size_t i = 0; i < inStates.size(); i++)
			  costs[i * cols + it->second] = lm_.Score(inStates[i], it->first, outStates[i * cols + it->second]);
	  };
	  pool.enqueue(call);
	}
  }
  algo::copy(costs.begin(), costs.end(), prob.begin());
}

State* LanguageModel::NewState() {
  return new LMState();
}

void LanguageModel::BeginSentenceState(State& state) {
  LMState& lmState = state.get<LMState>();
  lmState.GetStates().resize(1);
  lmState.GetStates()[0] = lm_.BeginSentenceState();
}

void LanguageModel::AssembleBeamState(const State& in,
							   const Beam& beam,
							   State& out) {

  const LMState& lmIn = in.get<LMState>();
  LMState& lmOut = out.get<LMState>();

  size_t cols = lmIn.GetStates().size() / beam.size();

  lmOut.GetStates().resize(beam.size());
  for(size_t i = 0; i < beam.size(); ++i)
	 lmOut.GetStates()[i] = lmIn.GetStates()[i * cols + beam[i]->GetWord()];
}

size_t LanguageModel::GetVocabSize() const {
  return lm_.size();
}

